#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <string>
#include <conio.h>

#include <stdio.h>
#include "utilities.h"
#include "SimpleGPU.cuh"
#include "SimpleCPU.h"
#include "SimplePCPU.h"
#include "AdvancedGPU.cuh"
#include "ComplexPCPU.h"

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	int iterations = 10000;
	int threads = 64;
	int width = 4096;
	int height = 4096;
	int size = width * height;


	int advancedWidth = width/8;
	int advancedHeight = height;
	int advancedSize = advancedWidth*advancedHeight;


	bool* map = new bool[size];
	bool* mapBuffer = new bool[size];

	generateMap(map, width, height);

	bool* mapGPU = new bool[size];
	std::copy(map, map + size, mapGPU);
	bool* mapCPU = new bool[size];
	std::copy(map, map + size, mapCPU);
	bool* mapPCPU = new bool[size];
	std::copy(map, map + size, mapPCPU);

	unsigned char* mapChar = new unsigned char[advancedSize];
	unsigned char* mapCharBuffer = new unsigned char[advancedSize];
	//generateMap(mapChar, advancedWidth, advancedHeight);
	copyBoolToCharMap(map, mapChar, advancedWidth, advancedHeight);

	if (!compareBoolToCharMap(map, mapChar, advancedWidth, advancedHeight)) {
		std::cout << "Copying map is incorrect" << std::endl;
	}

	//std::cout << "Bool map" << std::endl;
	//prettyPrint(map, width, height);
	//std::cout << "Char map" << std::endl;
	//prettyPrint(mapChar, advancedWidth, advancedHeight);


	//return 0;


	unsigned char* mapCharGPU = new unsigned char[advancedSize];
	std::copy(mapChar, mapChar + advancedSize, mapCharGPU);	
	
	unsigned char* mapCharPCPU = new unsigned char[advancedSize];
	std::copy(mapChar, mapChar + advancedSize, mapCharPCPU);

	auto start = std::chrono::high_resolution_clock::now();
	runEvaluateSimple(mapGPU, mapBuffer, width, height, iterations, threads);

	auto startCPU = std::chrono::high_resolution_clock::now();
	// iterationSerial(mapCPU, mapBuffer, iterations, height, width);

	auto startPCPU = std::chrono::high_resolution_clock::now();
	// iterationSimpleParallel(mapPCPU, mapBuffer, size, iterations, height, width);

	auto startAdvancedGPU = std::chrono::high_resolution_clock::now();

	runEvaluateAdvanced(mapCharGPU, mapCharBuffer, advancedWidth, advancedHeight, iterations, 1, threads);
	
	auto startComplexPCPU = std::chrono::high_resolution_clock::now();
	iterationComplexParallel(mapCharPCPU, mapCharBuffer, iterations, advancedHeight, advancedWidth, threads);
	auto stop = std::chrono::high_resolution_clock::now();

	std::cout << std::endl;

	std::cout << std::endl;
	auto durationGPU = std::chrono::duration_cast<std::chrono::milliseconds>(startCPU - start);
	auto durationCPU = std::chrono::duration_cast<std::chrono::milliseconds>(startPCPU - startCPU);
	auto durationPCPU = std::chrono::duration_cast<std::chrono::milliseconds>(startAdvancedGPU - startPCPU);
	auto durationAdvancedGPU = std::chrono::duration_cast<std::chrono::milliseconds>(startComplexPCPU - startPCPU);
	auto durationComplexPCPU = std::chrono::duration_cast<std::chrono::milliseconds>(stop - startComplexPCPU);
	std::cout << "gpu: " << durationGPU.count() << std::endl;
	//std::cout << "cpu: " << durationCPU.count() << std::endl;
	//std::cout << "pcpu: " << durationPCPU.count() << std::endl;
	std::cout << "advanced gpu: " << durationAdvancedGPU.count() << std::endl;
	std::cout << "complex pcpu: " << durationComplexPCPU.count() << std::endl;


	//if (!compareMap(mapCPU, mapGPU, width, height)) {
	//	std::cout << "GPU incorrect result map" << std::endl;
	//}
	//if (!compareMap(mapCPU, mapPCPU, width, height)) {
	//	std::cout << "PCPU incorrect result map" << std::endl;
	//}
	if (!compareBoolToCharMap(mapGPU, mapCharGPU, advancedWidth, advancedHeight)) {
		std::cout << "Advanced GPU incorrect result map" << std::endl;
	}	
	
	if (!compareBoolToCharMap(mapGPU, mapCharPCPU, advancedWidth, advancedHeight)) {
		std::cout << "Complex PCPU incorrect result map" << std::endl;
	}

	//std::cout << "Alive cells: " << aliveCells(mapGPU, width, height) << " " << aliveCells(mapCPU, width, height) << " " << aliveCells(mapPCPU, width, height) << std::endl;

	//prettyPrint(map, width, height);

	std::cout << "press any key to exit";
	//getch();

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
